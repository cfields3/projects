// Elapsed Real Time for input-5.txt: 0.726 seconds
// Type of GPU: nVidia RTX 2080 Super

/**
 * @file maxsum.cu
 * @author Christopher Fields (cwfields)
 *
 * Code for CSC 246 Assignment 5, Problem 3. Calculates
 * the maximum sum for a contiguous sequence of numbers
 * over an inputted list using a certain number of workers
 * as threads on a GPU serving as a co-processor.
 */

#include <stdio.h>
#include <stdbool.h>
#include <hip/hip_runtime.h>
#include <unistd.h>

// Input sequence of values.
int *vList;

// Number of values on the list.
int vCount = 0;

// Capacity of the list of values.
int vCap = 0;

// General function to report a failure and exit.
static void fail( char const *message ) {
  fprintf( stderr, "%s\n", message );
  exit( 1 );
}

// Print out a usage message, then exit.
static void usage() {
  printf( "usage: maxsum [report]\n" );
  exit( 1 );
}

// Read the list of values.
__host__ void readList() {
  // Set up initial list and capacity.
  vCap = 5;
  vList = (int *) malloc( vCap * sizeof( int ) );

  // Keep reading as many values as we can.
  int v;
  while ( scanf( "%d", &v ) == 1 ) {
    // Grow the list if needed.
    if ( vCount >= vCap ) {
      vCap *= 2;
      vList = (int *) realloc( vList, vCap * sizeof( int ) );
    }

    // Store the latest value in the next array slot.
    vList[ vCount++ ] = v;
  }
}

__global__ void checkSum( int vCount, bool report, int *devList, int *devSums ) {
  int idx = blockDim.x * blockIdx.x + threadIdx.x;

  if ( idx < vCount ) {
    // Initialize maxSum smallest value possible
    int maxSum = INT_MIN;

    // Calculate the maximum sum for specified worker
    int sum = 0;
    for( int i = idx; i < vCount; i++ ) {
      sum += devList[ i ];
      if ( sum > maxSum )
        maxSum = sum;
    }

    // Add the maximum sum found to the results array
    devSums[idx] = maxSum;

    // Print out a report for the work, if requested
    if ( report )
      printf( "I'm thread %d. The maximum sum I found is %d.\n", idx, maxSum );
  }
}

int main( int argc, char *argv[] ) {
  if ( argc < 1 || argc > 2 )
    usage();

  // If there's an argument, it better be "report"
  bool report = false;
  if ( argc == 2 ) {
    if ( strcmp( argv[ 1 ], "report" ) != 0 )
      usage();
    report = true;
  }

  readList();
  
  // Allocate memory on the device and copy over the list.
  int *devList = NULL;
  hipMalloc( (void **) &devList, vCount * sizeof(int) );

  // Copy the list over to the device.
  hipMemcpy( devList, vList, vCount * sizeof(int), hipMemcpyHostToDevice );

  // Allocate space on the device to hold the results.
  int *devSums = NULL;
  hipMalloc( (void **) &devSums, vCount * sizeof(int) );

  // Block and grid dimensions.
  int threadsPerBlock = 100;
  // Round up for the number of blocks we need.
  int blocksPerGrid = ( vCount + threadsPerBlock - 1 ) / threadsPerBlock;

  // Run our kernel on these block/grid dimensions
  checkSum<<<blocksPerGrid, threadsPerBlock>>>( vCount, report, devList, devSums );
  if ( hipGetLastError() != hipSuccess )
    fail( "Failure in CUDA kernel execution." );

  // Copy results back to the host
  int *sums = (int *) malloc( vCount * sizeof(int) );
  hipMemcpy( sums, devSums, vCount * sizeof(int), hipMemcpyDeviceToHost );

  // Compare the local largest products
  int maxSum = INT_MIN;
  for ( int i = 0; i < vCount; i++ ) {
    if ( sums[ i ] > maxSum )
      maxSum = sums[ i ];
  }

  // Report the final largest product
  printf( "Maximum Sum: %d\n", maxSum );

  // Free memory on the device and the host.
  free( vList );
  free( sums );
  hipFree( devList );
  hipFree( devSums );

  hipDeviceReset();

  return 0;
}
